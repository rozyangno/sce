#include "hip/hip_runtime.h"
#include <iostream>
#include <numeric>
#include <stdlib.h>
#include <stdio.h>
#include <cfloat>
#include <hiprand/hiprand_kernel.h>
#include <time.h>
#include <gsl/gsl_rng.h>
#include <gsl/gsl_randist.h>


static void
CheckCudaErrorAux (const char *file, unsigned line, const char *statement, hipError_t err)
{
	if (err == hipSuccess)
		return;
	std::cerr << statement << " returned " << hipGetErrorString (err) << "(" << err << ") at " << file << ":" << line
			<< std::endl;
	exit (EXIT_FAILURE);
}
#define CUDA_CHECK_RETURN(value) CheckCudaErrorAux(__FILE__,__LINE__, #value, value)


void fread_with_check(void *ptr, size_t size, size_t nmemb, FILE *stream)
{
	size_t nmemb_read = fread(ptr, size, nmemb, stream);
	if (nmemb_read == nmemb)
		return;
	printf("fread error");
	exit (EXIT_FAILURE);
}


#define MAX(a,b) ( (a) > (b) ? (a) : (b) )
#define MIN(a,b) ( (a) < (b) ? (a) : (b) )

#ifndef DIM
#define DIM 2
#endif

#define alpha0 0.5

typedef float real;
typedef long long longint;

int blockSize, blockCount, nWorker;

longint *I, *J;
double *P, *weights;
real *Y;
longint nn, ne, maxIter, nRepuSamp;
real eta0;
gsl_ran_discrete_t *gsl_de, *gsl_dn;
int bConstantEta;

longint *d_I, *d_J;
real *d_Y;
real *d_qsum;
real *d_qcount;
hiprandState *d_nnStates1, *d_nnStates2;
hiprandState *d_neStates;
gsl_ran_discrete_t *d_gsl_de, *d_gsl_dn;
double *d_gsl_de_F, *d_gsl_dn_F;
size_t *d_gsl_de_A, *d_gsl_dn_A;

real *d_Eq;
real *d_qsum_total;
real *d_qcount_total;

real alpha;

void
loadP (const char *fnameP, int bBinaryInput)
{
	FILE *fpP = fopen (fnameP, "r");
	if (bBinaryInput)
	{
		fread_with_check (&nn, sizeof(longint), 1, fpP);
		fread_with_check (&ne, sizeof(longint), 1, fpP);
		I = new longint[ne];
		J = new longint[ne];
		P = new double[ne];
		fread_with_check (I, sizeof(longint), ne, fpP);
		fread_with_check (J, sizeof(longint), ne, fpP);
		fread_with_check (P, sizeof(double), ne, fpP);
	}
	else
	{
		if (fscanf (fpP, "%lld %lld", &nn, &ne) != 2)
		{
			printf("Error in reading nn or ne!\n");
			exit(EXIT_FAILURE);
		}
		I = new longint[ne];
		J = new longint[ne];
		P = new double[ne];
		for (longint e = 0; e < ne; e++)
			if (fscanf (fpP, "%lld %lld %lg", I + e, J + e, P + e) != 3)
			{
				printf("Error in reading I, J, or P!\n");
				exit(EXIT_FAILURE);
			}
	}
	fclose (fpP);
}

void
loadWeights (const char *fnameWeights, int bBinaryInput)
{
	weights = new double[nn];
	if (strcmp (fnameWeights, "none") == 0)
	{
		for (longint i = 0; i < nn; i++)
			weights[i] = 1.0;
	}
	else
	{
		FILE *fpWeights = fopen (fnameWeights, "r");
		if (bBinaryInput)
			fread_with_check (weights, sizeof(double), nn, fpWeights);
		else
			for (longint i = 0; i < nn; i++)
				if (fscanf (fpWeights, "%lg", weights + i) != 1)
				{
					printf("Error in reading weights!\n");
					exit(EXIT_FAILURE);
				}
		fclose (fpWeights);
	}
}

void
loadY0 (const char *fnameY0, int bBinaryInput)
{
	Y = new real[nn * DIM];
	if (strcmp (fnameY0, "none") == 0)
	{
		srand (0);
		for (longint i = 0; i < nn; i++)
			for (longint d = 0; d < DIM; d++)
				Y[d + i * DIM] = rand () * 1e-4 / RAND_MAX;
	}
	else
	{
		FILE *fpY0 = fopen (fnameY0, "r");
		if (bBinaryInput)
			fread_with_check (Y, sizeof(real), nn * DIM, fpY0);
		else
			for (longint i = 0; i < nn; i++)
				if (fscanf (fpY0, "%f %f", Y + i * DIM, Y + i * DIM + 1) != 2)
				{
					printf("Error in reading Y0!\n");
					exit(EXIT_FAILURE);
				}
		fclose (fpY0);
	}
}

void
saveY (const char* fnameY)
{
	FILE *fpY = fopen (fnameY, "w+");
	for (longint i = 0; i < nn; i++)
	{
		for (longint d = 0; d < DIM; d++)
		{
			fprintf (fpY, "%.6f", Y[d + i * DIM]);
			if (d < DIM - 1)
				fprintf (fpY, " ");
		}
		fprintf (fpY, "\n");
	}

	fclose (fpY);
}

void
freeMemory ()
{
	delete[] I;
	delete[] J;
	delete[] P;
	delete[] Y;
	delete[] weights;
}

void
allocateDataAndCopy2Device ()
{
	CUDA_CHECK_RETURN(hipMalloc((void**)&d_Y, sizeof(real)*nn*DIM));
	CUDA_CHECK_RETURN(hipMalloc ((void** )&d_I, sizeof(longint) * ne));
	CUDA_CHECK_RETURN(hipMalloc ((void** )&d_J, sizeof(longint) * ne));

	CUDA_CHECK_RETURN(hipMemcpy(d_Y, Y, sizeof(real)*nn*DIM, hipMemcpyHostToDevice));
	CUDA_CHECK_RETURN(hipMemcpy (d_I, I, sizeof(longint) * ne, hipMemcpyHostToDevice));
	CUDA_CHECK_RETURN(hipMemcpy (d_J, J, sizeof(longint) * ne, hipMemcpyHostToDevice));

	real Eq = 1;
	CUDA_CHECK_RETURN(hipMalloc ((void** )&d_Eq, sizeof(real)));
	CUDA_CHECK_RETURN(hipMemcpy (d_Eq, &Eq, sizeof(real), hipMemcpyHostToDevice));

	CUDA_CHECK_RETURN(hipMalloc ((void ** )&d_qsum, nWorker * sizeof(real)));
	CUDA_CHECK_RETURN(hipMalloc ((void ** )&d_qcount, nWorker * sizeof(real)));
	CUDA_CHECK_RETURN(hipMalloc ((void ** )&d_qsum_total, sizeof(real)));
	CUDA_CHECK_RETURN(hipMalloc ((void ** )&d_qcount_total, sizeof(real)));

}

void
freeDataInDevice ()
{
	CUDA_CHECK_RETURN(hipFree (d_Y));
	CUDA_CHECK_RETURN(hipFree (d_I));
	CUDA_CHECK_RETURN(hipFree (d_J));
	CUDA_CHECK_RETURN(hipFree (d_qsum));
	CUDA_CHECK_RETURN(hipFree (d_qcount));

	CUDA_CHECK_RETURN(hipFree (d_nnStates1));
	CUDA_CHECK_RETURN(hipFree (d_nnStates2));
	CUDA_CHECK_RETURN(hipFree (d_neStates));
	CUDA_CHECK_RETURN(hipFree (d_gsl_de_A));
	CUDA_CHECK_RETURN(hipFree (d_gsl_de_F));
	CUDA_CHECK_RETURN(hipFree (d_gsl_de));
	CUDA_CHECK_RETURN(hipFree (d_gsl_dn_A));
	CUDA_CHECK_RETURN(hipFree (d_gsl_dn_F));
	CUDA_CHECK_RETURN(hipFree (d_gsl_dn));

	CUDA_CHECK_RETURN(hipFree (d_qsum_total));
	CUDA_CHECK_RETURN(hipFree (d_qcount_total));
}

__device__ size_t
my_curand_discrete (hiprandState *state, const gsl_ran_discrete_t *g)
{
	size_t c = 0;
	double u, f;
	u = hiprand_uniform (state);
	c = (u * (g->K));
	f = (g->F)[c];
	if (f == 1.0)
		return c;

	if (u < f)
	{
		return c;
	}
	else
	{
		return (g->A)[c];
	}
}

__global__ void
setupCURANDKernel (hiprandState *nnStates1, hiprandState *nnStates2, hiprandState *neStates)
{
	longint workerIdx = (longint) (blockIdx.x * blockDim.x + threadIdx.x);
	hiprand_init (314159, /* the seed */
				 workerIdx, /* the sequence number */
				 0, /* not use the offset */
				 &nnStates1[workerIdx]);
	hiprand_init (314159 + 1, /* the seed */
				 workerIdx, /* the sequence number */
				 0, /* not use the offset */
				 &nnStates2[workerIdx]);
	hiprand_init (271828, /* the seed */
				 workerIdx, /* the sequence number */
				 0, /* not use the offset */
				 &neStates[workerIdx]);
}

__global__ void
assembleGSLKernel (gsl_ran_discrete_t *d_gsl_de, size_t *d_gsl_de_A, double *d_gsl_de_F, gsl_ran_discrete_t *d_gsl_dn,
				   size_t *d_gsl_dn_A, double *d_gsl_dn_F)
{
	d_gsl_de->A = d_gsl_de_A;
	d_gsl_de->F = d_gsl_de_F;
	d_gsl_dn->A = d_gsl_dn_A;
	d_gsl_dn->F = d_gsl_dn_F;
}

void
setupDiscreteDistribution ()
{
	CUDA_CHECK_RETURN(hipMalloc ((void ** )&d_nnStates1, blockCount * blockSize * sizeof(hiprandState)));
	CUDA_CHECK_RETURN(hipMalloc ((void ** )&d_nnStates2, blockCount * blockSize * sizeof(hiprandState)));
	CUDA_CHECK_RETURN(hipMalloc ((void ** )&d_neStates, blockCount * blockSize * sizeof(hiprandState)));
	setupCURANDKernel <<<blockCount, blockSize>>> (d_nnStates1, d_nnStates2, d_neStates);

	gsl_rng_env_setup ();
	gsl_de = gsl_ran_discrete_preproc (ne, P);
	gsl_dn = gsl_ran_discrete_preproc (nn, weights);
	CUDA_CHECK_RETURN(hipMalloc ((void ** )&d_gsl_de, sizeof(gsl_ran_discrete_t)));
	CUDA_CHECK_RETURN(hipMalloc ((void ** )&d_gsl_dn, sizeof(gsl_ran_discrete_t)));
	CUDA_CHECK_RETURN(hipMalloc ((void ** )&d_gsl_de_A, sizeof(size_t) * ne));
	CUDA_CHECK_RETURN(hipMalloc ((void ** )&d_gsl_de_F, sizeof(double) * ne));
	CUDA_CHECK_RETURN(hipMalloc ((void ** )&d_gsl_dn_A, sizeof(size_t) * nn));
	CUDA_CHECK_RETURN(hipMalloc ((void ** )&d_gsl_dn_F, sizeof(double) * nn));
	CUDA_CHECK_RETURN(hipMemcpy (d_gsl_de, gsl_de, sizeof(gsl_ran_discrete_t), hipMemcpyHostToDevice));
	CUDA_CHECK_RETURN(hipMemcpy (d_gsl_de_A, gsl_de->A, sizeof(size_t) * ne, hipMemcpyHostToDevice));
	CUDA_CHECK_RETURN(hipMemcpy (d_gsl_de_F, gsl_de->F, sizeof(double) * ne, hipMemcpyHostToDevice));
	CUDA_CHECK_RETURN(hipMemcpy (d_gsl_dn, gsl_dn, sizeof(gsl_ran_discrete_t), hipMemcpyHostToDevice));
	CUDA_CHECK_RETURN(hipMemcpy (d_gsl_dn_A, gsl_dn->A, sizeof(size_t) * nn, hipMemcpyHostToDevice));
	CUDA_CHECK_RETURN(hipMemcpy (d_gsl_dn_F, gsl_dn->F, sizeof(double) * nn, hipMemcpyHostToDevice));
	assembleGSLKernel <<<1, 1>>> (d_gsl_de, d_gsl_de_A, d_gsl_de_F, d_gsl_dn, d_gsl_dn_A, d_gsl_dn_F);
	gsl_ran_discrete_free (gsl_de);
	gsl_ran_discrete_free (gsl_dn);
}

__global__ void
sceUpdateYKernel (hiprandState *nnStates1, hiprandState *nnStates2, hiprandState *neStates, gsl_ran_discrete_t* d_gsl_dn,
					gsl_ran_discrete_t* d_gsl_de, real *Y, longint *I, longint *J, real *d_Eq, real *qsum, real *qcount,
					longint nn, longint ne, real eta, longint nRepuSamp, real nsq, real attrCoef, real alpha)
{
	int workerIdx = blockIdx.x * blockDim.x + threadIdx.x;
	real dY[DIM];
	real c = 1.0 / ((*d_Eq) * nsq);
	qsum[workerIdx] = 0.0;
	qcount[workerIdx] = 0.0;

	real repuCoef = 2 * c / nRepuSamp * nsq;
	for (longint r = 0; r < nRepuSamp + 1; r++)
	{
		longint k, l;
		if (r == 0)
		{
			longint e = (longint) (my_curand_discrete (neStates + workerIdx, d_gsl_de) % ne);
			k = I[e];
			l = J[e];
		}
		else
		{
			k = (longint) (my_curand_discrete (nnStates1 + workerIdx, d_gsl_dn) % nn);
			l = (longint) (my_curand_discrete (nnStates2 + workerIdx, d_gsl_dn) % nn);
		}

		if (k == l)
			continue;

		longint lk = k * DIM;
		longint ll = l * DIM;
		real dist2 = 0.0;
		for (longint d = 0; d < DIM; d++)
		{
			dY[d] = Y[d + lk] - Y[d + ll];
			dist2 += dY[d] * dY[d];
		}
		real q = 1.0 / (1 + dist2);

		real g;
		if (r == 0)
			g = -attrCoef * q;
		else
			g = repuCoef * q * q;

		for (longint d = 0; d < DIM; d++)
		{
			real gain = eta * g * dY[d];
			Y[d + lk] += gain;
			Y[d + ll] -= gain;

		}
		qsum[workerIdx] += r==0 ? alpha * q : (1-alpha) * q;
		qcount[workerIdx] += r==0 ? alpha : (1-alpha);
	}
}

__global__ void
resetQsumQCountTotalKernel (real *d_qsum_total, real *d_qcount_total)
{
	(*d_qsum_total) = 0.0;
	(*d_qcount_total) = 0;
}

template<typename T>
	__global__ void
	reduceSumArrayKernel (T *array, int n, T* arraySum)
	{
		T sum = 0;
		for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < n; i += blockDim.x * gridDim.x)
		{
			sum += array[i];
		}
		atomicAdd (arraySum, sum);
	}

__global__ void
updateEqKernel (real *d_Eq, real *d_qsum_total, real* d_qcount_total, real nsq)
{
	(*d_Eq) = ((*d_Eq) * nsq + (*d_qsum_total)) / (nsq + (*d_qcount_total));
}

void
sce ()
{
	//hipSetDevice (0);
	hipDeviceReset ();

	nWorker = blockSize * blockCount;
	real nsq = (real) nn * (nn - 1);

	real Psum = 0.0;
	for (longint e = 0; e < ne; e++)
		Psum += P[e];
	for (longint e = 0; e < ne; e++)
		P[e] /= Psum;

	real wsum = 0.0;
	for (longint i=0; i<nn; i++)
		wsum += weights[i];
	for (longint i=0; i<nn; i++)
		weights[i] /= wsum;

	allocateDataAndCopy2Device ();

	setupDiscreteDistribution ();


	for (longint iter = 0; iter < maxIter; iter++)
	{
		real eta;
		if (bConstantEta)
			eta = eta0;
		else {
			eta = eta0 * (1 - (real) iter / (maxIter - 1));
			eta = MAX(eta, eta0 * 1e-4);
		}

		real alpha_effective = alpha;

		real attrCoef = 2;
		sceUpdateYKernel <<<blockCount, blockSize>>> (d_nnStates1, d_nnStates2, d_neStates, d_gsl_dn, d_gsl_de, d_Y,
														d_I, d_J, d_Eq, d_qsum, d_qcount, nn, ne, eta, nRepuSamp, nsq,
														attrCoef, alpha_effective);

		resetQsumQCountTotalKernel <<<1, 1>>> (d_qsum_total, d_qcount_total);
		reduceSumArrayKernel <<<16, 128>>> (d_qsum, nWorker, d_qsum_total);
		reduceSumArrayKernel <<<16, 128>>> (d_qcount, nWorker, d_qcount_total);
		updateEqKernel <<<1, 1>>> (d_Eq, d_qsum_total, d_qcount_total, nsq);

		if (iter % MAX(1, maxIter / 1000) == 0)
		{
			printf ("%cOptimizing progress: %.3lf%%", 13, (real) iter / (real) maxIter * 100);
			fflush (stdout);
		}
	}

	CUDA_CHECK_RETURN(hipMemcpy(Y, d_Y, sizeof(real)*nn*DIM, hipMemcpyDeviceToHost));

	freeDataInDevice ();
}


int
main (int argc, char **argv)
{
	printf ("Usage: sce bBinaryInput P_file Y_file weights_file Y0_file maxIter eta0 nRepuSamp blockSize blockCount alpha bConstantEta\n");
	int bBinaryInput = atoi(argv[1]);
	const char *fnameP = argv[2];
	const char *fnameY = argv[3];
	const char *fnameWeights = argv[4];
	const char *fnameY0 = argv[5];
	maxIter = atoi (argv[6]);
	eta0 = atof (argv[7]);
	nRepuSamp = atoi (argv[8]);
	blockSize = atoi (argv[9]);
	blockCount = atoi (argv[10]);
	alpha = atof(argv[11]);
	bConstantEta = atoi(argv[12]);

	printf ("maxIter=%lld, eta0=%f, nRepuSamp=%lld, blockSize=%d, blockCount=%d, bConstantEta=%d\n", maxIter, eta0, nRepuSamp,
			blockSize, blockCount, bConstantEta);

	loadP (fnameP, bBinaryInput);
	loadWeights (fnameWeights, bBinaryInput);
	loadY0 (fnameY0, bBinaryInput);

	clock_t start = clock ();
	sce ();
	clock_t end = clock ();
	float seconds = (float) (end - start) / CLOCKS_PER_SEC;
	printf ("\nSCE used %.2f seconds\n", seconds);

	saveY (fnameY);

	freeMemory ();

	printf ("Done.\n");

	return 0;
}

